/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <array>
#include <memory>

#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/conv_op.h"
#include "paddle/fluid/operators/layout_utils.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/phi/backends/dynload/hipDNN.h"
#include "paddle/phi/kernels/gpudnn/conv_cudnn_frontend.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = platform::DataLayout;
using helper = phi::CudnnFrontendConvHelper;

template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

namespace {
cudnn_frontend::Operation MakeDreluOp(hipdnnDataType_t dtype,
                                      cudnn_frontend::Tensor const& dy_desc,
                                      cudnn_frontend::Tensor const& x_desc,
                                      cudnn_frontend::Tensor const& dx_desc) {
  auto op_desc = cudnn_frontend::PointWiseDescBuilder()
                     .setMode(CUDNN_POINTWISE_RELU_BWD)
                     .setComputeType(dtype)
                     .build();
  auto op = cudnn_frontend::OperationBuilder(
                CUDNN_BACKEND_OPERATION_POINTWISE_DESCRIPTOR)
                .setdyDesc(dy_desc)
                .setxDesc(x_desc)
                .setdxDesc(dx_desc)
                .setpwDesc(op_desc)
                .build();
  VLOG(6) << op.describe();
  return op;
}

cudnn_frontend::Operation MakeBnbwdweightOp(
    hipdnnDataType_t dtype,
    cudnn_frontend::Tensor const& x_desc,
    cudnn_frontend::Tensor const& mean_desc,
    cudnn_frontend::Tensor const& invstd_desc,
    cudnn_frontend::Tensor const& bn_scale_desc,
    cudnn_frontend::Tensor const& dy_desc,
    cudnn_frontend::Tensor const& dbn_bias_desc,
    cudnn_frontend::Tensor const& dbn_scale_desc,
    cudnn_frontend::Tensor const& eq_dy_scale_desc,
    cudnn_frontend::Tensor const& eq_x_scale_desc,
    cudnn_frontend::Tensor const& eqbias_desc) {
  auto op =
      cudnn_frontend::OperationBuilder(
          CUDNN_BACKEND_OPERATION_BN_BWD_WEIGHTS_DESCRIPTOR)
          .setComputeType(dtype)
          .setxDesc(x_desc)
          .setSavedMeanAndInvVar(mean_desc, invstd_desc)
          .setScale(bn_scale_desc)
          .setdyDesc(dy_desc)
          .setEqScalesAndBias(eq_dy_scale_desc, eq_x_scale_desc, eqbias_desc)
          .setDScaleAndDBias(dbn_scale_desc, dbn_bias_desc)
          .build();
  VLOG(6) << op.describe();
  return op;
}
}  // namespace

template <typename T>
class FusedDgradDreluBnBwdWeightOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    using U = BatchNormParamType<T>;
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();
    PADDLE_ENFORCE_EQ(
        dev_ctx.GetComputeCapability(),
        80,
        phi::errors::PreconditionNotMet("This op only supports Ampere devices, "
                                        "but got compute capability: %d.",
                                        dev_ctx.GetComputeCapability()));
    auto cudnn_version = platform::DnnVersion();
    PADDLE_ENFORCE_GE(cudnn_version,
                      8800,
                      phi::errors::PreconditionNotMet(
                          "This op only supports CUDNN version >= 8800, "
                          "but got %d.",
                          cudnn_version));
    // Attributes
    bool fuse_shortcut = ctx.Attr<bool>("fuse_shortcut");
    bool fuse_dual = ctx.Attr<bool>("fuse_dual");
    bool fuse_add = ctx.Attr<bool>("fuse_add");
    // required input variables
    const Tensor* dy_tensor = ctx.Input<Tensor>("dY");
    const Tensor* w_tensor = ctx.Input<Tensor>("W");
    const Tensor* bn1_mean_tensor = ctx.Input<Tensor>("BN1_mean");
    const Tensor* bn1_inv_std_tensor = ctx.Input<Tensor>("BN1_inv_std");
    const Tensor* bn1_scale_tensor = ctx.Input<Tensor>("BN1_scale");
    const Tensor* bn1_bias_tensor = ctx.Input<Tensor>("BN1_bias");
    const Tensor* bn1_x_tensor = ctx.Input<Tensor>("BN1_X");
    // dispensable inputs
    const Tensor* relu_x_tensor = nullptr;
    const Tensor* bn2_mean_tensor = nullptr;
    const Tensor* bn2_inv_std_tensor = nullptr;
    const Tensor* bn2_scale_tensor = nullptr;
    const Tensor* bn2_bias_tensor = nullptr;
    const Tensor* bn2_x_tensor = nullptr;
    const Tensor* dx_branch_tensor = nullptr;
    if (fuse_shortcut) {
      relu_x_tensor = ctx.Input<Tensor>("Relu_X");
    }
    if (fuse_dual) {
      bn2_mean_tensor = ctx.Input<Tensor>("BN2_mean");
      bn2_inv_std_tensor = ctx.Input<Tensor>("BN2_inv_std");
      bn2_scale_tensor = ctx.Input<Tensor>("BN2_scale");
      bn2_bias_tensor = ctx.Input<Tensor>("BN2_bias");
      bn2_x_tensor = ctx.Input<Tensor>("BN2_X");
    }
    if (fuse_add) {
      dx_branch_tensor = ctx.Input<Tensor>("dX_branch");
    }

    // required output variables
    Tensor* dx_tensor = ctx.Output<Tensor>("dX");
    Tensor* bn1_dgamma_tensor = ctx.Output<Tensor>("BN1_dGamma");
    Tensor* bn1_dbeta_tensor = ctx.Output<Tensor>("BN1_dBeta");
    Tensor* bn1_dBeta_tensor = ctx.Output<Tensor>("BN1_eqscale_dy");
    Tensor* bn1_eqscale_x_tensor = ctx.Output<Tensor>("BN1_eqscale_x");
    Tensor* bn1_eqbias_tensor = ctx.Output<Tensor>("BN1_eqbias");
    dx_tensor->mutable_data<T>(ctx.GetPlace());
    bn1_dgamma_tensor->mutable_data<U>(ctx.GetPlace());
    bn1_dbeta_tensor->mutable_data<U>(ctx.GetPlace());
    bn1_dBeta_tensor->mutable_data<U>(ctx.GetPlace());
    bn1_eqscale_x_tensor->mutable_data<U>(ctx.GetPlace());
    bn1_eqbias_tensor->mutable_data<U>(ctx.GetPlace());

    // dispensable outputs
    Tensor* bn2_dgamma_tensor = nullptr;
    Tensor* bn2_dbeta_tensor = nullptr;
    Tensor* bn2_eqscale_dy_tensor = nullptr;
    Tensor* bn2_eqscale_x_tensor = nullptr;
    Tensor* bn2_eqbias_tensor = nullptr;
    if (fuse_dual) {
      bn2_dgamma_tensor = ctx.Output<Tensor>("BN2_dGamma");
      bn2_dbeta_tensor = ctx.Output<Tensor>("BN2_dBeta");
      bn2_eqscale_dy_tensor = ctx.Output<Tensor>("BN2_eqscale_dy");
      bn2_eqscale_x_tensor = ctx.Output<Tensor>("BN2_eqscale_x");
      bn2_eqbias_tensor = ctx.Output<Tensor>("BN2_eqbias");

      bn2_dgamma_tensor->mutable_data<U>(ctx.GetPlace());
      bn2_dbeta_tensor->mutable_data<U>(ctx.GetPlace());
      bn2_eqscale_dy_tensor->mutable_data<U>(ctx.GetPlace());
      bn2_eqscale_x_tensor->mutable_data<U>(ctx.GetPlace());
      bn2_eqbias_tensor->mutable_data<U>(ctx.GetPlace());
    }
    // transform filter to NHWC layout
    Tensor w_tensor_transformed(w_tensor->dtype());
    using Context = phi::GPUContext;
    ResizeToChannelLast<Context, T>(ctx, w_tensor, &w_tensor_transformed);
    TransToChannelLast<Context, T>(ctx, w_tensor, &w_tensor_transformed);
    // deal with strides, dilations and paddings
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    const std::string padding_algorithm =
        ctx.Attr<std::string>("padding_algorithm");
    // update padding and dilation
    auto in_dims = bn1_x_tensor->dims();
    auto filter_dims = w_tensor_transformed.dims();
    framework::DDim in_data_dims = slice_ddim(in_dims, 1, in_dims.size() - 1);
    framework::DDim filter_data_dims =
        slice_ddim(filter_dims, 1, filter_dims.size() - 1);
    std::vector<int> ksize = phi::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(
        &paddings, &dilations, padding_algorithm, in_data_dims, strides, ksize);
    int data_dim = strides.size();  // 2d or 3d
    std::vector<int64_t> pre_padding(data_dim, 0);
    std::vector<int64_t> post_padding(data_dim, 0);
    for (size_t i = 0; i < data_dim; ++i) {
      pre_padding[i] = static_cast<int64_t>(paddings[2 * i]);
      post_padding[i] = static_cast<int64_t>(paddings[2 * i + 1]);
    }
    // get handles
    auto handle = dev_ctx.cudnn_handle();
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    // build tensor descriptors
    hipdnnTensorFormat_t layout_format = HIPDNN_TENSOR_NHWC;
    auto tensor_format = platform::ToCudnnDataType(
        framework::TransToProtoVarType(dy_tensor->dtype()));
    auto tensor_format_math = HIPDNN_DATA_FLOAT;
    auto compute_dtype = HIPDNN_DATA_FLOAT;

    // get dims in CUDNN manner: [N, C, H, W]
    auto dim_x = platform::TransformDimOrder(phi::vectorize<int64_t>(in_dims));
    auto dim_filt =
        platform::TransformDimOrder(phi::vectorize<int64_t>(filter_dims));
    auto dim_y =
        platform::TransformDimOrder(phi::vectorize<int64_t>(dy_tensor->dims()));
    std::vector<int64_t> dim_scale(dim_x.size(), 1);
    dim_scale[1] = dim_x[1];  //  [1, C, 1, 1]

    std::vector<void*> data_ptrs;
    std::vector<int64_t> uids;
    int64_t uid = 100;

    // Build tensor descriptors
    // dgrad inputs
    auto dy_desc = helper::GetGeneralTensorDescriptor(
        dim_y, layout_format, ++uid, 16, tensor_format);
    data_ptrs.push_back(const_cast<T*>(dy_tensor->data<T>()));
    uids.push_back(uid);

    auto w_desc = helper::GetGeneralTensorDescriptor(
        dim_filt, layout_format, ++uid, 16, tensor_format);
    data_ptrs.push_back(const_cast<T*>(w_tensor_transformed.data<T>()));
    uids.push_back(uid);

    // dBN1 inputs
    auto bn1_mean_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(const_cast<U*>(bn1_mean_tensor->data<U>()));
    uids.push_back(uid);

    auto bn1_inv_std_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(const_cast<U*>(bn1_inv_std_tensor->data<U>()));
    uids.push_back(uid);

    auto bn1_scale_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(const_cast<U*>(bn1_scale_tensor->data<U>()));
    uids.push_back(uid);

    auto bn1_bias_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(const_cast<U*>(bn1_bias_tensor->data<U>()));
    uids.push_back(uid);

    auto bn1_x_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format);
    data_ptrs.push_back(const_cast<T*>(bn1_x_tensor->data<T>()));
    uids.push_back(uid);

    // dBN2 inputs
    auto bn2_mean_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(const_cast<U*>(bn2_mean_tensor->data<U>()));
      uids.push_back(uid);
    }

    auto bn2_inv_std_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(const_cast<U*>(bn2_inv_std_tensor->data<U>()));
      uids.push_back(uid);
    }

    auto bn2_scale_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(const_cast<U*>(bn2_scale_tensor->data<U>()));
      uids.push_back(uid);
    }

    auto bn2_bias_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(const_cast<U*>(bn2_bias_tensor->data<U>()));
      uids.push_back(uid);
    }

    auto bn2_x_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format);
    if (fuse_dual) {
      data_ptrs.push_back(const_cast<T*>(bn2_x_tensor->data<T>()));
      uids.push_back(uid);
    }

    // shortcut input
    auto relu_x_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format);
    if (fuse_shortcut) {
      data_ptrs.push_back(const_cast<T*>(relu_x_tensor->data<T>()));
      uids.push_back(uid);
    }

    // fuse_add inputs
    auto dx_branch_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format);
    if (fuse_add) {
      data_ptrs.push_back(const_cast<T*>(dx_branch_tensor->data<T>()));
      uids.push_back(uid);
    }

    // virtual outputs
    auto dx_dgrad_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_add0 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_add1 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_mul1 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_add2 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_mul2 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto final_bitmask_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_dual_add1 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_dual_mul1 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_dual_add2 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);
    auto after_dual_mul2 = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format_math, true);

    // drelu outputs
    auto dx_desc = helper::GetGeneralTensorDescriptor(
        dim_x, layout_format, ++uid, 16, tensor_format);
    data_ptrs.push_back(dx_tensor->data<T>());
    uids.push_back(uid);

    // dBN1 outputs
    auto bn1_dgamma_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(bn1_dgamma_tensor->data<U>());
    uids.push_back(uid);

    auto bn1_dbeta_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(bn1_dbeta_tensor->data<U>());
    uids.push_back(uid);

    auto bn1_eqscale_dy_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(bn1_dBeta_tensor->data<U>());
    uids.push_back(uid);

    auto bn1_eqscale_x_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(bn1_eqscale_x_tensor->data<U>());
    uids.push_back(uid);

    auto bn1_eqbias_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    data_ptrs.push_back(bn1_eqbias_tensor->data<U>());
    uids.push_back(uid);

    // dBN2 outputs
    auto bn2_dgamma_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(bn2_dgamma_tensor->data<U>());
      uids.push_back(uid);
    }
    auto bn2_dbeta_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(bn2_dbeta_tensor->data<U>());
      uids.push_back(uid);
    }
    auto bn2_eqscale_dy_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(bn2_eqscale_dy_tensor->data<U>());
      uids.push_back(uid);
    }
    auto bn2_eqscale_x_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(bn2_eqscale_x_tensor->data<U>());
      uids.push_back(uid);
    }
    auto bn2_eqbias_desc = helper::GetGeneralTensorDescriptor(
        dim_scale, layout_format, ++uid, 16, tensor_format_math);
    if (fuse_dual) {
      data_ptrs.push_back(bn2_eqbias_tensor->data<U>());
      uids.push_back(uid);
    }

    // build ops
    std::vector<cudnn_frontend::Operation const*> ops;
    // make dgrad op
    std::vector<int64_t> stride_int64 = helper::GetInt64Array(strides);
    std::vector<int64_t> dilation_int64 = helper::GetInt64Array(dilations);
    auto conv_desc = cudnn_frontend::ConvDescBuilder()
                         .setComputeType(HIPDNN_DATA_FLOAT)
                         .setMathMode(HIPDNN_CROSS_CORRELATION)
                         .setSpatialDimCount(data_dim)
                         .setSpatialStride(data_dim, stride_int64.data())
                         .setPrePadding(data_dim, pre_padding.data())
                         .setPostPadding(data_dim, post_padding.data())
                         .setDilation(data_dim, dilation_int64.data())
                         .build();
    VLOG(6) << conv_desc.describe();

    auto dgrad_op =
        cudnn_frontend::OperationBuilder(
            CUDNN_BACKEND_OPERATION_CONVOLUTION_BACKWARD_DATA_DESCRIPTOR)
            .setdyDesc(dy_desc)
            .setwDesc(w_desc)
            .setdxDesc(dx_dgrad_desc)
            .setcDesc(conv_desc)
            .setAlpha(1.0f)
            .setBeta(0.0f)
            .build();
    VLOG(6) << dgrad_op.describe();
    ops.push_back(&dgrad_op);

    cudnn_frontend::Tensor* p_drelu_input_desc = &dx_dgrad_desc;
    auto add0_op = helper::MakePointwiseOp(CUDNN_POINTWISE_ADD,
                                           compute_dtype,
                                           dx_dgrad_desc,
                                           dx_branch_desc,
                                           after_add0);
    if (fuse_add) {
      ops.push_back(&add0_op);
      p_drelu_input_desc = &after_add0;
    }
    // make pointwise nodes
    auto add1_op = helper::MakePointwiseOp(CUDNN_POINTWISE_ADD,
                                           compute_dtype,
                                           bn1_x_desc,
                                           bn1_mean_desc,
                                           after_add1,
                                           1.0,
                                           -1.0);
    ops.push_back(&add1_op);

    auto mul1_op = helper::MakePointwiseOp(CUDNN_POINTWISE_MUL,
                                           compute_dtype,
                                           after_add1,
                                           bn1_inv_std_desc,
                                           after_mul1);
    ops.push_back(&mul1_op);

    auto mul2_op = helper::MakePointwiseOp(CUDNN_POINTWISE_MUL,
                                           compute_dtype,
                                           after_mul1,
                                           bn1_scale_desc,
                                           after_mul2);
    ops.push_back(&mul2_op);

    auto add2_op = helper::MakePointwiseOp(CUDNN_POINTWISE_ADD,
                                           compute_dtype,
                                           after_mul2,
                                           bn1_bias_desc,
                                           after_add2);
    ops.push_back(&add2_op);

    auto dual_add1_op = helper::MakePointwiseOp(CUDNN_POINTWISE_ADD,
                                                compute_dtype,
                                                bn2_x_desc,
                                                bn2_mean_desc,
                                                after_dual_add1,
                                                1.0,
                                                -1.0);
    if (fuse_dual) ops.push_back(&dual_add1_op);

    auto dual_mul1_op = helper::MakePointwiseOp(CUDNN_POINTWISE_MUL,
                                                compute_dtype,
                                                after_dual_add1,
                                                bn2_inv_std_desc,
                                                after_dual_mul1);
    if (fuse_dual) ops.push_back(&dual_mul1_op);

    auto dual_mul2_op = helper::MakePointwiseOp(CUDNN_POINTWISE_MUL,
                                                compute_dtype,
                                                after_dual_mul1,
                                                bn2_scale_desc,
                                                after_dual_mul2);
    if (fuse_dual) ops.push_back(&dual_mul2_op);

    auto dual_add2_op = helper::MakePointwiseOp(CUDNN_POINTWISE_ADD,
                                                compute_dtype,
                                                after_dual_mul2,
                                                bn2_bias_desc,
                                                after_dual_add2);
    if (fuse_dual) ops.push_back(&dual_add2_op);

    cudnn_frontend::Tensor* p_bmask_input_desc =
        fuse_shortcut ? &relu_x_desc : &after_dual_add2;
    auto bmask_add_op = helper::MakePointwiseOp(CUDNN_POINTWISE_ADD,
                                                compute_dtype,
                                                after_add2,
                                                *p_bmask_input_desc,
                                                final_bitmask_desc);
    if (fuse_shortcut || fuse_dual) ops.push_back(&bmask_add_op);

    cudnn_frontend::Tensor* p_drelu_bmask_desc =
        (fuse_shortcut || fuse_dual) ? &final_bitmask_desc : &after_add2;
    auto drelu_op = MakeDreluOp(
        compute_dtype, *p_drelu_input_desc, *p_drelu_bmask_desc, dx_desc);
    ops.push_back(&drelu_op);

    auto bn_bwd_weight_op = MakeBnbwdweightOp(compute_dtype,
                                              bn1_x_desc,
                                              bn1_mean_desc,
                                              bn1_inv_std_desc,
                                              bn1_scale_desc,
                                              dx_desc,
                                              bn1_dbeta_desc,
                                              bn1_dgamma_desc,
                                              bn1_eqscale_dy_desc,
                                              bn1_eqscale_x_desc,
                                              bn1_eqbias_desc);
    ops.push_back(&bn_bwd_weight_op);

    auto dual_bn_bwd_weight_op = MakeBnbwdweightOp(compute_dtype,
                                                   bn2_x_desc,
                                                   bn2_mean_desc,
                                                   bn2_inv_std_desc,
                                                   bn2_scale_desc,
                                                   dx_desc,
                                                   bn2_dbeta_desc,
                                                   bn2_dgamma_desc,
                                                   bn2_eqscale_dy_desc,
                                                   bn2_eqscale_x_desc,
                                                   bn2_eqbias_desc);
    if (fuse_dual) ops.push_back(&dual_bn_bwd_weight_op);

    // build op graph
    auto op_graph = cudnn_frontend::OperationGraphBuilder()
                        .setHandle(handle)
                        .setOperationGraph(ops.size(), ops.data())
                        .build();
    VLOG(6) << op_graph.describe();

    auto plan = helper::GetPlanByHeuristics(std::move(op_graph), handle);
    VLOG(6) << "Plan tag: " << plan.getTag();

    auto workspace_size = plan.getWorkspaceSize();
    VLOG(4) << plan.describe() << " requires workspace " << workspace_size;

    helper::ExecutePlan(handle,
                        &workspace_handle,
                        &data_ptrs,
                        &uids,
                        plan.get_raw_desc(),
                        workspace_size);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fused_dgrad_drelu_bnbwdweight,
    ops::FusedDgradDreluBnBwdWeightOpKernel<paddle::platform::float16>);
